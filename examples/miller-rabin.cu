#include "hip/hip_runtime.h"
#include <moderngpu/kernel_reduce.hxx>
#include <hiprand-done-right/curanddr.hxx>

using ulong = unsigned long;

int main(int argc, char**argv) {
  uint size = 1;
  int input_n = 17;

  if(argc > 1)
    sscanf(argv[1], "%d", &input_n);

  /* Write n-1 as 2^k * d, where k is maximal. In other words, find
   * the rightmost 1 in the bitwise representation of n-1; the number
   * with just that bit set is 2^k. */
  int nm1 = input_n - 1;
  int two_to_k = nm1 & -nm1;
  int dd = nm1 / two_to_k;
  assert(dd*two_to_k == nm1);

  mgpu::standard_context_t context;

  mgpu::mem_t<int> result(1, context);
  mgpu::transform_reduce(
    [=]__device__(uint index) {
      auto randoms = curanddr::uniforms<1>(uint4{0,0,0,0},
                                           index);
      int base_a = 2 + randoms[0]*(input_n-4);
      
      uint xx = 1;
      for(int ii = 0; ii < dd; ++ii)
        xx = xx*base_a % input_n;

      if(xx == 1 or xx == input_n - 1) {
        return 0;
      }

      for(int power = two_to_k; power > 2; power /= 2) {
        xx = xx*xx % input_n;

        if(xx == 1) {
          return 1;
        }
        if(xx == input_n-1)
          return 0;
      }

      return 1;
    },
    size,
    result.data(),
    mgpu::plus_t<int>(),
    context);

  if(mgpu::from_mem(result)[0] > 0)
    printf("Composite\n");
  else
    printf("Probably prime\n");
  return 0;
}
